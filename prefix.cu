#include"stdio.h"
#include"hip/hip_runtime.h"
__global__ void esort(int *a,int *b)
{
int tid=threadIdx.x;
int sum=0;
for(int i=0;i<=tid;i++)
{
sum=sum+a[i];
}

b[tid]=sum;
}

int main()
{
int a[10]={1,2,3,4,5,6,7,8,9,10};
int *deva,*devb;
size_t bytes=10*sizeof(int);
hipMalloc((void**)&deva,bytes);
hipMalloc((void**)&devb,bytes);
hipMemcpy(deva,a,bytes,hipMemcpyHostToDevice);
esort<<<1,10>>>(deva,devb);
hipMemcpy(a,devb,bytes,hipMemcpyDeviceToHost);
printf("sorted array\n");
for(int i=0;i<10;i++)
{
printf("%d\t",a[i]);
}
return 0;
}


