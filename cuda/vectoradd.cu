#include "hip/hip_runtime.h"
#include "stdio.h"
#define N 3
__global__ void vectoradd(int *c, int *a, int *b)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    c[id] = a[id] + b[id];
}
int main()
{
    int a[N], b[N], c[N];
    int *da, *db, *dc;
    hipMalloc((void **)&da, N * sizeof(int));
    hipMalloc((void **)&db, N * sizeof(int));
    hipMalloc((void **)&dc, N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        a[i] = i + 1;
        b[i] = i + 1;
    }
    hipMemcpy(da, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(db, b, N * sizeof(int), hipMemcpyHostToDevice);
    vectoradd<<<2, (N / 2) + 1>>>(dc, da, db);
    hipMemcpy(c, dc, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%d + %d = %d\n", a[i], b[i], c[i]);
    }
    return 0;
}
