#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"
__global__ void fibonacci(int *a)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;

    for (int i = 0; i < 2; i++)
    {
        a[id] = round(pow(1.618, id) / 2.2360);
    }
}
int main()
{
    int *a;
    int *dev_arr;
    hipMalloc((void **)&dev_arr, 50 * sizeof(int));
    a = (int *)malloc(50 * sizeof(int));
    fibonacci<<<2, 25>>>(dev_arr);
    hipMemcpy(a, dev_arr, 50 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 50; i++)
    {
        printf("%d ", a[i]);
    }
    printf("\n");
    return 0;
}