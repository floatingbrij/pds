#include "hip/hip_runtime.h"
#include "stdio.h"
__global__ void prime(int *dev_a)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x + 1;
    int n = dev_a[100];
    int count = 0;
    int a[100];
    int sum1 = 0;
    for (int k = 1; k < n; k++)
    {
        if (id % k)
        {
            a[count] = k;
            count++;
            sum1 += a[count];
        }
    }
    if (id == sum1)
    {
        dev_a[id] = id;
    }
    else
    {
        dev_a[id] = 0;
    }
}
int main()
{
    int arr[100];
    int n = 100;
    int *dev_a;
    int a[100];
    for (int i = 0; i < n; i++)
    {
        a[i] = 0;
    }
    // n = (int *)malloc(49 * sizeof(int));
    hipMalloc((void **)&dev_a, n * sizeof(int));
    // cudaMalloc((void **)&dev_r, 49 * sizeof(int));
    hipMemcpy(dev_a, a, n * sizeof(int), hipMemcpyHostToDevice);
    // cudaMemcpy(dev_r, res, 49 * sizeof(int), cudaMemcpyHostToDevice);
    prime<<<10, 10>>>(dev_a);
    hipMemcpy(a, dev_a, n * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < n; i++)
    {
        if (a[i] != 0)
        {
            printf("%d  ", a[i]);
        }
    }
    return 0;
}
