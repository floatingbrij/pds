#include "hip/hip_runtime.h"
#include "stdio.h"
#define N 10
__global__ void enumsort(int *deva, int *devn)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int i, count = 0;
    for (i = 0; i < N; i++)
    {
        if ((deva[i] <= deva[id]) && (i != id))
        {
            count++;
        }
        devn[count] = deva[id];
    }
}
int main()
{
    int a[] = {19, 500, 29, 306, 65, 38, 1, 59, 254, 41};
    int *deva, *n, *devn;
    n = (int *)malloc(N * sizeof(int));
    hipMalloc((void **)&deva, N * sizeof(int));
    hipMalloc((void **)&devn, N * sizeof(int));
    hipMemcpy(deva, a, N * sizeof(int), hipMemcpyHostToDevice);
    enumsort<<<2, 5>>>(deva, devn);
    hipMemcpy(n, devn, N * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < N; i++)
    {
        printf("%d ", n[i]);
    }
    return 0;
}
