#include "hip/hip_runtime.h"
#include "stdio.h"
#include "math.h"
__global__ void amstrong(int *dev_a, int *sum)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int n = dev_a[id];
    int s = 0;
    int z = n;
    int y = n;
    int c = 0;
    while (n != 0)
    {
        n = n / 10;
        c++;
    }
    while (y != 0)
    {
        int r = y % 10;
        s += pow(r, c);
        y = y / 10;
    }
    if (s == z)
    {
        sum[id] = 1;
    }
}
int main()
{
    int arr[1000], res[1000];
    int *dev_a, *sum, *n;
    n = (int *)malloc(1000 * sizeof(int));
    for (int i = 0; i < 1000; i++)
    {
        arr[i] = i + 1;
        res[i] = 0;
    }
    hipMalloc((void **)&dev_a, 1000 * sizeof(int));
    hipMalloc((void **)&sum, 1000 * sizeof(int));
    hipMemcpy(dev_a, arr, 1000 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(sum, res, 1000 * sizeof(int), hipMemcpyHostToDevice);
    amstrong<<<2, 500>>>(dev_a, sum);
    hipMemcpy(n, sum, 1000 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 1000; i++)
    {
        if (n[i] == 1)
        {
            printf("%d ", arr[i]);
        }
    }
    return 0;
}
