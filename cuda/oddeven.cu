
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define N 100
__global__ void oddevensort(int *a)
{
    int ix = threadIdx.x;
    for (int i = 0; i < N / 2; i++)
    {
        if (ix < N - 1 && ix % 2 != 0)
        {
            int t = a[ix + 1];
            a[ix + 1] = max(t, a[ix]);
            a[ix] = min(t, a[ix]);
        }
        __syncthreads();
        if (ix < N - 1 && ix % 2 == 0)
        {
            int t = a[ix + 1];
            a[ix + 1] = max(t, a[ix]);
            a[ix] = min(t, a[ix]);
        }
        __syncthreads();
    }
}
int main()
{
    int *arr;
    int *dev_arr;
    arr = (int *)malloc(N * sizeof(int));
    hipMalloc((void **)&dev_arr, N * sizeof(int));
    int ct = N;
    printf("Input array is :\n");
    for (int i = 0; i < N; i++)
    {
        arr[i] = ct;
        ct--;
        printf("%d ", arr[i]);
    }
    printf("\n");
    hipMemcpy(dev_arr, arr, N * sizeof(int), hipMemcpyHostToDevice);
    oddevensort<<<1, N>>>(dev_arr);
    hipMemcpy(arr, dev_arr, N * sizeof(int), hipMemcpyDeviceToHost);
    hipDeviceReset();
    printf("Output array is :\n");
    for (int i = 0; i < N; i++)
    {
        printf("%d ", arr[i]);
    }
    printf("\n");
    hipFree(dev_arr);
    free(arr);
}
