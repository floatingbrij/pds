#include "hip/hip_runtime.h"
#include "stdio.h"
#include "string.h"
__global__ void utol(char *deva, int n)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if (deva[id] >= 65 && deva[id] <= 90)
    {
        deva[id] = deva[id] + 32;
    }
}
int main()
{
    char a[100];
    printf("\n enter string");
    scanf("%[^\n]%*c", a);
    int len = strlen(a);
    char *deva;
    hipMalloc((void **)&deva, len * sizeof(char));
    hipMemcpy(deva, a, len * sizeof(char), hipMemcpyHostToDevice);
    utol<<<2, (len / 2) + 1>>>(deva, len);
    hipMemcpy(&a, deva, len * sizeof(char), hipMemcpyDeviceToHost);
    printf("string:%s", a);
    return 0;
}