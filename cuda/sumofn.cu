#include "hip/hip_runtime.h"
#include "stdio.h"

__global__ void sumofnat(int *dev_arr, int *res)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    // res+=dev_arr[id];
    atomicAdd(res, dev_arr[id]);
}
int main()
{
    int arr[100], *n;
    int *dev_arr, *res;
    for (int i = 0; i < 100; i++)
    {
        arr[i] = i + 1;
    }
    n = (int *)malloc(sizeof(int));
    hipMalloc((void **)&dev_arr, 100 * sizeof(int));
    hipMalloc((void **)&res, sizeof(int));
    hipMemcpy(dev_arr, arr, 100 * sizeof(int), hipMemcpyHostToDevice);
    sumofnat<<<2, 50>>>(dev_arr, res);
    hipMemcpy(n, res, sizeof(int), hipMemcpyDeviceToHost);
    printf("Result:%d\n", *n);
    return 0;
}