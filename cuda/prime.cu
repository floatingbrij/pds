#include "hip/hip_runtime.h"
#include "stdio.h"
__global__ void prime(int *dev_a, int *dev_r)
{
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    int n = dev_a[id];
    if (n * n < 49 && dev_r[id] != 1)
    {
        for (int i = id + 1; i < 49; i++)
        {
            if (dev_a[i] % n == 0)
            {
                dev_r[i] = 1;
            }
        }
    }
}
int main()
{
    int arr[49], res[49];
    for (int i = 0; i < 49; i++)
    {
        arr[i] = i + 2;
        res[i] = 0;
    }
    int *dev_a, *dev_r, *n;
    n = (int *)malloc(49 * sizeof(int));
    hipMalloc((void **)&dev_a, 49 * sizeof(int));
    hipMalloc((void **)&dev_r, 49 * sizeof(int));
    hipMemcpy(dev_a, arr, 49 * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_r, res, 49 * sizeof(int), hipMemcpyHostToDevice);
    prime<<<2, 25>>>(dev_a, dev_r);
    hipMemcpy(n, dev_r, 49 * sizeof(int), hipMemcpyDeviceToHost);
    for (int i = 0; i < 49; i++)
    {
        if (n[i] == 0)
        {
            printf("%d  ", arr[i]);
        }
    }
    return 0;
}
