#include "stdio.h"
#include "hip/hip_runtime.h"
__constant__ float carray[100];
__global__ void vadd(float *darray )
{
int index;
index=blockIdx.x*blockDim.x+threadIdx.x;

for(int i=0;i<100;i++)
{
darray[index]=darray[index]+carray[i];
}
return;
}

int main()
{
int size=3200;
size_t bytes=size*sizeof(float);
float rarray[3200];
float *darray;
float harray[100];
hipMalloc((void**)&darray,bytes);
hipMemset(darray,0,bytes);

for(int i=0;i<100;i++)
{
harray[i]=i+1;
}

hipMemcpyToSymbol(HIP_SYMBOL(carray),harray,sizeof(float)*100);

hipEvent_t start,stop;
hipEventCreate(&start);
hipEventCreate(&stop);
hipEventRecord(start);

vadd<<<size/64,64>>>(darray);

hipEventRecord(stop);
hipEventSynchronize(stop);
float t=0;
hipEventElapsedTime(&t,start,stop);
hipMemcpy(rarray,darray,bytes,hipMemcpyDeviceToHost);


printf("RESULT ARRAY printed only from index 0 to 9\n");
for(int j=0;j<10;j++)
{
printf("%f \n",rarray[j]);
}
printf("\nTime taken using constant memory :%f",t);

hipFree(darray);
return 0;
}
                                                                                                                           

