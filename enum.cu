#include"stdio.h"
#include"hip/hip_runtime.h"
__global__ void enum_sort(int *a,int *b)
{
int t=threadIdx.x;
int count=0;
for(int i=0;i<10;i++)
{
if(a[i]<a[t])
 {
  count++;
 }
}
b[count]=a[t];
}

int main()
{
int a[10]={44,33,22,11,33,77,99,66,88,100};
int *dev_a,*dev_b;
size_t bytes=10*sizeof(int);
hipMalloc((void**)&dev_a,bytes);
hipMalloc((void**)&dev_b,bytes);
hipMemcpy(dev_a,a,bytes,hipMemcpyHostToDevice);
enum_sort<<<1,10>>>(dev_a,dev_b);
hipMemcpy(a,dev_b,bytes,hipMemcpyDeviceToHost);
printf("SORTED ARRAY IS\n");
for(int i=0;i<10;i++)
{
printf("%d\t",a[i]);
}
printf("\n");
return 0;
}






